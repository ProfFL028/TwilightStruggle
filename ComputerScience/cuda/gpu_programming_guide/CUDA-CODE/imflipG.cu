#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <iostream>
#include <ctype.h>


#define DATAMB(bytes)			(bytes/1024/1024)
#define DATABW(bytes,timems)	((float)bytes/(timems * 1.024*1024.0*1024.0))

typedef unsigned char uch;
typedef unsigned long ul;
typedef unsigned int  ui;

uch *TheImg, *CopyImg;					// Where images are stored in CPU
uch *GPUImg, *GPUCopyImg, *GPUResult;	// Where images are stored in GPU

struct ImgProp{
	int Hpixels;
	int Vpixels;
	uch HeaderInfo[54];
	ul Hbytes;
} ip;

#define	IPHB		ip.Hbytes
#define	IPH			ip.Hpixels
#define	IPV			ip.Vpixels
#define	IMAGESIZE	(IPHB*IPV)
#define	IMAGEPIX	(IPH*IPV)



// Kernel that flips the given image vertically
// each thread only flips a single pixel (R,G,B)
__global__
void Vflip(uch *ImgDst, uch *ImgSrc, ui Hpixels, ui Vpixels)
{
	ui ThrPerBlk = blockDim.x;
	ui MYbid = blockIdx.x;
	ui MYtid = threadIdx.x;
	ui MYgtid = ThrPerBlk * MYbid + MYtid;

	ui BlkPerRow = (Hpixels + ThrPerBlk - 1) / ThrPerBlk;  // ceil
	ui RowBytes = (Hpixels * 3 + 3) & (~3);
	ui MYrow = MYbid / BlkPerRow;
	ui MYcol = MYgtid - MYrow*BlkPerRow*ThrPerBlk;
	if (MYcol >= Hpixels) return;			// col out of range
	ui MYmirrorrow = Vpixels - 1 - MYrow;
	ui MYsrcOffset = MYrow       * RowBytes;
	ui MYdstOffset = MYmirrorrow * RowBytes;
	ui MYsrcIndex = MYsrcOffset + 3 * MYcol;
	ui MYdstIndex = MYdstOffset + 3 * MYcol;

	// swap pixels RGB   @MYcol , @MYmirrorcol
	ImgDst[MYdstIndex] = ImgSrc[MYsrcIndex];
	ImgDst[MYdstIndex + 1] = ImgSrc[MYsrcIndex + 1];
	ImgDst[MYdstIndex + 2] = ImgSrc[MYsrcIndex + 2];
}


// Kernel that flips the given image horizontally
// each thread only flips a single pixel (R,G,B)
__global__
void Hflip(uch *ImgDst, uch *ImgSrc, ui Hpixels)
{
	ui ThrPerBlk = blockDim.x;
	ui MYbid = blockIdx.x;
	ui MYtid = threadIdx.x;
	ui MYgtid = ThrPerBlk * MYbid + MYtid;

	ui BlkPerRow = (Hpixels + ThrPerBlk -1 ) / ThrPerBlk;  // ceil
	ui RowBytes = (Hpixels * 3 + 3) & (~3);
	ui MYrow = MYbid / BlkPerRow;
	ui MYcol = MYgtid - MYrow*BlkPerRow*ThrPerBlk;
	if (MYcol >= Hpixels) return;			// col out of range
	ui MYmirrorcol = Hpixels - 1 - MYcol;
	ui MYoffset = MYrow * RowBytes;
	ui MYsrcIndex = MYoffset + 3 * MYcol;
	ui MYdstIndex = MYoffset + 3 * MYmirrorcol;

	// swap pixels RGB   @MYcol , @MYmirrorcol
	ImgDst[MYdstIndex] = ImgSrc[MYsrcIndex];
	ImgDst[MYdstIndex + 1] = ImgSrc[MYsrcIndex + 1];
	ImgDst[MYdstIndex + 2] = ImgSrc[MYsrcIndex + 2];
}


// Kernel that copies an image from one part of the
// GPU memory (ImgSrc) to another (ImgDst)
__global__
void PixCopy(uch *ImgDst, uch *ImgSrc, ui FS)
{
	ui ThrPerBlk = blockDim.x;
	ui MYbid = blockIdx.x;
	ui MYtid = threadIdx.x;
	ui MYgtid = ThrPerBlk * MYbid + MYtid;

	if (MYgtid > FS) return;				// outside the allocated memory
	ImgDst[MYgtid] = ImgSrc[MYgtid];
}


/*
// helper function that wraps CUDA API calls, reports any error and exits
void chkCUDAErr(cudaError_t error_id)
{
	if (error_id != CUDA_SUCCESS)
	{
		printf("CUDA ERROR :::%\n", cudaGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
}
*/


// Read a 24-bit/pixel BMP file into a 1D linear array.
// Allocate memory to store the 1D image and return its pointer.
uch *ReadBMPlin(char* fn)
{
	static uch *Img;
	FILE* f = fopen(fn, "rb");
	if (f == NULL){	printf("\n\n%s NOT FOUND\n\n", fn);	exit(EXIT_FAILURE); }

	uch HeaderInfo[54];
	fread(HeaderInfo, sizeof(uch), 54, f); // read the 54-byte header
	// extract image height and width from header
	int width = *(int*)&HeaderInfo[18];			ip.Hpixels = width;
	int height = *(int*)&HeaderInfo[22];		ip.Vpixels = height;
	int RowBytes = (width * 3 + 3) & (~3);		ip.Hbytes = RowBytes;
	//save header for re-use
	memcpy(ip.HeaderInfo, HeaderInfo,54);
	printf("\n Input File name: %17s  (%u x %u)   File Size=%u", fn, 
			ip.Hpixels, ip.Vpixels, IMAGESIZE);
	// allocate memory to store the main image (1 Dimensional array)
	Img  = (uch *)malloc(IMAGESIZE);
	if (Img == NULL) return Img;      // Cannot allocate memory
	// read the image from disk
	fread(Img, sizeof(uch), IMAGESIZE, f);
	fclose(f);
	return Img;
}


// Write the 1D linear-memory stored image into file.
void WriteBMPlin(uch *Img, char* fn)
{
	FILE* f = fopen(fn, "wb");
	if (f == NULL){ printf("\n\nFILE CREATION ERROR: %s\n\n", fn); exit(1); }
	//write header
	fwrite(ip.HeaderInfo, sizeof(uch), 54, f);
	//write data
	fwrite(Img, sizeof(uch), IMAGESIZE, f);
	printf("\nOutput File name: %17s  (%u x %u)   File Size=%u", fn, ip.Hpixels, ip.Vpixels, IMAGESIZE);
	fclose(f);
}


int main(int argc, char **argv)
{
	char Flip = 'H';
	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime; // GPU code run times
	hipError_t cudaStatus, cudaStatus2;
	hipEvent_t time1, time2, time3, time4;
	char InputFileName[255], OutputFileName[255], ProgName[255];
	ui BlkPerRow, ThrPerBlk=256, NumBlocks, GPUDataTransfer;
	hipDeviceProp_t GPUprop;
	ul SupportedKBlocks, SupportedMBlocks, MaxThrPerBlk;		char SupportedBlocks[100];


	strcpy(ProgName, "imflipG");
	switch (argc){
	case 5:  ThrPerBlk=atoi(argv[4]);
	case 4:  Flip = toupper(argv[3][0]);
	case 3:  strcpy(InputFileName, argv[1]);
			 strcpy(OutputFileName, argv[2]);
			 break;
	default: printf("\n\nUsage:   %s InputFilename OutputFilename [V/H/C/T] [ThrPerBlk]", ProgName);
			 printf("\n\nExample: %s Astronaut.bmp Output.bmp", ProgName);
			 printf("\n\nExample: %s Astronaut.bmp Output.bmp H", ProgName);
			 printf("\n\nExample: %s Astronaut.bmp Output.bmp V  128",ProgName);
			 printf("\n\nH=horizontal flip, V=vertical flip, T=Transpose, C=copy image\n\n");
			 exit(EXIT_FAILURE);
	}
	if ((Flip != 'V') && (Flip != 'H') && (Flip != 'C') && (Flip != 'T')) {
		printf("Invalid flip option '%c'. Must be 'V','H', 'T', or 'C'... \n", Flip);
		exit(EXIT_FAILURE);
	}
	if ((ThrPerBlk < 32) || (ThrPerBlk > 1024)) {
		printf("Invalid ThrPerBlk option '%u'. Must be between 32 and 1024. \n", ThrPerBlk);
		exit(EXIT_FAILURE);
	}

	// Create CPU memory to store the input and output images
	TheImg = ReadBMPlin(InputFileName); // Read the input image if memory can be allocated
	if (TheImg == NULL){
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}
	CopyImg = (uch *)malloc(IMAGESIZE);
	if (CopyImg == NULL){
		free(TheImg);
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		printf("\nNo CUDA Device is available\n");
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exit(EXIT_FAILURE);
	}
	hipGetDeviceProperties(&GPUprop, 0);
	SupportedKBlocks = (ui)GPUprop.maxGridSize[0] * (ui)GPUprop.maxGridSize[1] * (ui)GPUprop.maxGridSize[2] / 1024;
	SupportedMBlocks = SupportedKBlocks / 1024;
	sprintf(SupportedBlocks, "%u %c", (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks, (SupportedMBlocks >= 5) ? 'M' : 'K');
	MaxThrPerBlk = (ui)GPUprop.maxThreadsPerBlock;

	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);

	hipEventRecord(time1, 0);		// Time stamp at the start of the GPU transfer
	// Allocate GPU buffer for the input and output images
	cudaStatus = hipMalloc((void**)&GPUImg, IMAGESIZE);
	cudaStatus2 = hipMalloc((void**)&GPUCopyImg, IMAGESIZE);
	if ((cudaStatus != hipSuccess) || (cudaStatus2 != hipSuccess)){
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory");
		exit(EXIT_FAILURE);
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(GPUImg, TheImg, IMAGESIZE, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!");
		exit(EXIT_FAILURE);
	}

	hipEventRecord(time2, 0);		// Time stamp after the CPU --> GPU tfr is done
	
	//dim3 dimBlock(ThrPerBlk);
	//dim3 dimGrid(ip.Hpixels*BlkPerRow);

	BlkPerRow = (IPH + ThrPerBlk -1 ) / ThrPerBlk;
	NumBlocks = IPV*BlkPerRow; 
	switch (Flip){
		case 'H': Hflip <<< NumBlocks, ThrPerBlk >>> (GPUCopyImg, GPUImg, IPH);
				  GPUResult = GPUCopyImg;
				  GPUDataTransfer = 2*IMAGESIZE;
				  break;
		case 'V': Vflip <<< NumBlocks, ThrPerBlk >>> (GPUCopyImg, GPUImg, IPH, IPV);
				  GPUResult = GPUCopyImg;
				  GPUDataTransfer = 2*IMAGESIZE;
				  break;
		case 'T': Hflip <<< NumBlocks, ThrPerBlk >>> (GPUCopyImg, GPUImg, IPH);
				  Vflip <<< NumBlocks, ThrPerBlk >>> (GPUImg, GPUCopyImg, IPH, IPV);
				  GPUResult = GPUImg;
				  GPUDataTransfer = 4*IMAGESIZE;
				  break;
		case 'C': NumBlocks = (IMAGESIZE+ThrPerBlk-1) / ThrPerBlk;
				  PixCopy <<< NumBlocks, ThrPerBlk >>> (GPUCopyImg, GPUImg, IMAGESIZE);
				  GPUResult = GPUCopyImg;
				  GPUDataTransfer = 2*IMAGESIZE;
				  break;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\nhipDeviceSynchronize returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time3, 0);

	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(CopyImg, GPUResult, IMAGESIZE, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy GPU to CPU  failed!");
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time4, 0);

	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	cudaStatus = hipDeviceSynchronize();
	//checkError(cudaGetLastError());	// screen for errors in kernel launches
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n Program failed after cudaDeviceSynchronize()!");
		free(TheImg);
		free(CopyImg);
		exit(EXIT_FAILURE);
	}
	WriteBMPlin(CopyImg, OutputFileName);		// Write the flipped image back to disk
	printf("\n\n--------------------------------------------------------------------------\n");
	printf("%s    ComputeCapab=%d.%d  [max %s blocks; %d thr/blk] \n", 
			GPUprop.name, GPUprop.major, GPUprop.minor, SupportedBlocks, MaxThrPerBlk);
	printf("--------------------------------------------------------------------------\n");
	printf("%s %s %s %c %u   [%u BLOCKS, %u BLOCKS/ROW]\n", ProgName, InputFileName, OutputFileName,
			Flip, ThrPerBlk, NumBlocks, BlkPerRow);
	printf("--------------------------------------------------------------------------\n");
	printf("CPU->GPU Transfer   =%7.2f ms  ...  %4d MB  ...  %6.2f GB/s\n", tfrCPUtoGPU, DATAMB(IMAGESIZE), DATABW(IMAGESIZE, tfrCPUtoGPU));
	printf("Kernel Execution    =%7.2f ms  ...  %4d MB  ...  %6.2f GB/s\n", kernelExecutionTime, DATAMB(GPUDataTransfer), DATABW(GPUDataTransfer, kernelExecutionTime));
	printf("GPU->CPU Transfer   =%7.2f ms  ...  %4d MB  ...  %6.2f GB/s\n", tfrGPUtoCPU, DATAMB(IMAGESIZE), DATABW(IMAGESIZE, tfrGPUtoCPU));
	printf("--------------------------------------------------------------------------\n");
	printf("Total time elapsed  =%7.2f ms       %4d MB  ...  %6.2f GB/s\n", totalTime, DATAMB((2 * IMAGESIZE + GPUDataTransfer)), DATABW((2 * IMAGESIZE + GPUDataTransfer), totalTime));
	printf("--------------------------------------------------------------------------\n\n");

	// Deallocate CPU, GPU memory and destroy events.
	hipFree(GPUImg);
	hipFree(GPUCopyImg);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(TheImg);
		free(CopyImg);
		exit(EXIT_FAILURE);
	}
	free(TheImg);
	free(CopyImg);
	return(EXIT_SUCCESS);
}



